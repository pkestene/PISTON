#include "hip/hip_runtime.h"
/*
Copyright (c) 2012, Los Alamos National Security, LLC
All rights reserved.
Copyright 2012. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Authors: Ollie Lo and Christopher Sewell, ollie@lanl.gov and csewell@lanl.gov
*/

#ifdef __APPLE__
  #include <GL/glew.h>
  #include <OpenGL/OpenGL.h>
  #include <GLUT/glut.h>
#else
  #include <GL/glew.h>
  #include <GL/glut.h>
  #include <GL/gl.h>
#endif

#include <QtGui>
#include <QObject>

#ifdef USE_INTEROP
#include <cuda_gl_interop.h>
#endif

#include <piston/piston_math.h> 
#include <piston/choose_container.h>

#define SPACE thrust::device_space_tag
using namespace piston;

#include <piston/implicit_function.h>
#include <piston/image3d.h>
#include <piston/marching_cube.h>
#include <piston/util/tangle_field.h>
#include <piston/util/plane_field.h>
#include <piston/util/sphere_field.h>
#include <piston/threshold_geometry.h>

#include <sys/time.h>
#include <stdio.h>
#include <math.h>

#include "glwindow.h"


//==========================================================================
/*! 
    Variable declarations
*/
//==========================================================================

//! Variables for timing the framerate
struct timeval begin, end, diff;
int frameCount;

//! Tangle field and marching cube operator
tangle_field<SPACE>* tangle;
marching_cube<tangle_field<SPACE>, tangle_field<SPACE> > *isosurface;

//! Vertices, normals, and colors for output
thrust::host_vector<float4> vertices;
thrust::host_vector<float3> normals;
thrust::host_vector<float4> colors;

//! Camera and UI variables
float cameraFOV;
int gridSize;
bool wireframe;
float minIsovalue, maxIsovalue;

//! Vertex buffer objects used by CUDA interop
#ifdef USE_INTEROP
  GLuint vboBuffers[4];  struct hipGraphicsResource* vboResources[4];
#endif


//==========================================================================
/*! 
    Constructor for GLWindow class

    \fn	GLWindow::GLWindow
*/
//==========================================================================
GLWindow::GLWindow(QWidget *parent)
    : QGLWidget(QGLFormat(QGL::SampleBuffers), parent)
{
    // Start the QT callback timer
    setFocusPolicy(Qt::StrongFocus);
    timer = new QTimer(this);
    connect(timer, SIGNAL(timeout()), this, SLOT(updateGL()));
    timer->start(1);
}


//==========================================================================
/*! 
    Create the tangle field and the marching cubes operator

    \fn	GLWindow::initializeGL
*/
//==========================================================================
void GLWindow::initializeGL()
{
    // Initialize camera and UI variables
    qrot.set(0,0,0,1);
    frameCount = 0;
    gridSize = 64;
    cameraFOV = 60.0f;
    minIsovalue = 31.0f;
    maxIsovalue = 500.0f;
    wireframe = false;

    // Set up basic OpenGL state and lighting
    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);
    float white[] = { 0.8, 0.8, 0.8, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { 0.0, 0.0, gridSize*1.5, 1.0 };
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);
    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
    glEnable(GL_COLOR_MATERIAL);

    // Initialize CUDA interop if it is being used
    #ifdef USE_INTEROP
      glewInit();
      cudaGLSetGLDevice(0);
    #endif

    // Create the tangle field and marching cube operator instances
    tangle = new tangle_field<SPACE>(gridSize, gridSize, gridSize);
    isosurface = new marching_cube<tangle_field<SPACE>,  tangle_field<SPACE> >(*tangle, *tangle, 0.2f);

    // Compute the isosurface of the tangle field 
    (*isosurface)();

    // If using interop, generate the vertex buffer objects to be shared between CUDA and OpenGL
    #ifdef USE_INTEROP
      int numPoints = thrust::distance(isosurface->vertices_begin(), isosurface->vertices_end());
      glGenBuffers(3, vboBuffers);
      for (int i=0; i<3; i++)
      {
        unsigned int bufferSize = (i == 2) ? numPoints*sizeof(float3) : numPoints*sizeof(float4);
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[i]);
        glBufferData(GL_ARRAY_BUFFER, bufferSize, 0, GL_DYNAMIC_DRAW);
      }
      glBindBuffer(GL_ARRAY_BUFFER, 0);
      for (int i=0; i<3; i++)
      {
        hipGraphicsGLRegisterBuffer(&(vboResources[i]), vboBuffers[i], cudaGraphicsMapFlagsWriteDiscard);   
        isosurface->vboResources[i] = vboResources[i];
      }
      isosurface->minIso = minIsovalue;  isosurface->maxIso = maxIsovalue;  isosurface->useInterop = true;  isosurface->vboSize = numPoints;
    #endif

    // Enable OpenGL state for vertex, normal, and color arrays
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
}


//==========================================================================
/*! 
    Update the graphics

    \fn	GLWindow::paintGL
*/
//==========================================================================
void GLWindow::paintGL()
{
    // Stop the QT callback timer
    timer->stop();

    // Start timing this interval
    if (frameCount == 0) gettimeofday(&begin, 0);

    // Set up the OpenGL state
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    if (wireframe) glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    else glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    // Set up the projection and modelview matrices for the view
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(cameraFOV, 1.0f, 1.0f, gridSize*4.0f);
    glMatrixMode(GL_MODELVIEW);
    glPushMatrix();
    glLoadIdentity();
    gluLookAt(0.0f, 0.0f, 4.0f, 0.0f, 0.0f, 0.0f, 0.0f, 1.0f, 0.0f);

    // Set up the current rotation and translation
    qrot.getRotMat(rotationMatrix);
    glMultMatrixf(rotationMatrix);  

    // Compute the isosurface of the tangle field
    (*isosurface)();

    // If using interop, render the vertex buffer objects; otherwise, render the arrays
    #ifdef USE_INTEROP
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[0]);
      glVertexPointer(4, GL_FLOAT, 0, 0);

      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[2]);
      glNormalPointer(GL_FLOAT, 0, 0);

      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[1]);
      glColorPointer(4, GL_FLOAT, 0, 0);

      glDrawArrays(GL_TRIANGLES, 0, isosurface->vboSize);
    #else
      vertices.assign(isosurface->vertices_begin(), isosurface->vertices_end());
      normals.assign(isosurface->normals_begin(), isosurface->normals_end());
      colors.assign(thrust::make_transform_iterator(isosurface->scalars_begin(), color_map<float>(31.0f, 500.0f)),
                    thrust::make_transform_iterator(isosurface->scalars_end(), color_map<float>(31.0f, 500.0f)));
      glColorPointer(4, GL_FLOAT, 0, &colors[0]);
      glNormalPointer(GL_FLOAT, 0, &normals[0]);
      glVertexPointer(4, GL_FLOAT, 0, &vertices[0]);
      glDrawArrays(GL_TRIANGLES, 0, vertices.size());
    #endif

    // Pop this OpenGL view matrix
    glPopMatrix();

    // Periodically output the framerate
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    frameCount++;
    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    if (seconds > 0.5f)
    {
      char title[256];
      sprintf(title, "Marching Cube, fps: %2.2f", float(frameCount)/seconds);
      std::cout << title << std::endl;
      seconds = 0.0f;
      frameCount = 0;
    }

    // Restart the QT callback timer
    timer->start(1);
}


//==========================================================================
/*! 
    Handle window resize event

    \fn	GLWindow::resizeGL
*/
//==========================================================================
void GLWindow::resizeGL(int width, int height)
{
    glViewport(0, 0, width, height);
}


//==========================================================================
/*! 
    Handle mouse press event

    \fn	GLWindow::mousePressEvent
*/
//==========================================================================
void GLWindow::mousePressEvent(QMouseEvent *event)
{
    lastPos = event->pos();
}


//==========================================================================
/*! 
    Handle mouse move event to rotate, translate, or zoom

    \fn	GLWindow::mouseMoveEvent
*/
//==========================================================================
void GLWindow::mouseMoveEvent(QMouseEvent *event)
{
    int dx = event->x() - lastPos.x();
    int dy = event->y() - lastPos.y();

    // Rotate or zoom the view
    if (event->buttons() & Qt::LeftButton)
    {
      Quaternion newRotX;
      newRotX.setEulerAngles(-0.2f*dx*3.14159f/180.0f, 0.0f, 0.0f);
      qrot.mul(newRotX);

      Quaternion newRotY;
      newRotY.setEulerAngles(0.0f, 0.0f, -0.2f*dy*3.14159f/180.0f);
      qrot.mul(newRotY);
    }
    else if (event->buttons() & Qt::RightButton)
    {
      cameraFOV += dy/20.0f;
    }
    lastPos = event->pos();
}


//==========================================================================
/*! 
    Handle keyboard input event

    \fn	GLWindow::keyPressEvent
*/
//==========================================================================
void GLWindow::keyPressEvent(QKeyEvent *event)
{
   // Toggle wireframe mode
   if ((event->key() == 'w') || (event->key() == 'W'))
       wireframe = !wireframe;
}


