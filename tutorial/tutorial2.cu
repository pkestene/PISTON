#include "hip/hip_runtime.h"
/*
Copyright (c) 2012, Los Alamos National Security, LLC
All rights reserved.
Copyright 2012. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Christopher Sewell, csewell@lanl.gov
This simulation is based on the method by Matt Sottile described here: http://syntacticsalt.com/2011/03/10/functional-flocks/
*/

#ifdef __APPLE__
  #include <GL/glew.h>
  #include <OpenGL/OpenGL.h>
  #include <GLUT/glut.h>
#else
  #include <GL/glew.h>
  #include <GL/glut.h>
  #include <GL/gl.h>
#endif

#include <QtGui>
#include <QObject>

#ifdef USE_INTEROP
#include <cuda_gl_interop.h>
#endif

#include <sys/time.h>
#include <stdio.h>
#include <math.h>

#include <vtkSphereSource.h>
#include <vtkArrowSource.h>
#include <vtkPolyData.h>
#include <vtkCellArray.h>
#include <vtkCellData.h>
#include <vtkDoubleArray.h>
#include <vtkFloatArray.h>
#include <vtkPoints.h>
#include <vtkPolyData.h>
#include <vtkPolyDataNormals.h>
#include <vtkPointData.h>
#include <vtkSmartPointer.h>
#include <vtkXMLPolyDataReader.h>
#include <vtkTriangleFilter.h>
#include <vtkPolyDataNormals.h>

#include <piston/piston_math.h> 
#include <piston/choose_container.h>
#include <piston/hsv_color_map.h>

#define SPACE thrust::device_space_tag
using namespace piston;

#include "flock_sim.h"
#include "glyph.h"
#include "glwindow.h"

//! Number of boids
#define INPUT_SIZE 1024


//==========================================================================
/*! 
    Variable declarations
*/
//==========================================================================

//! Variables for timing the framerate
struct timeval begin, end, diff;
int frameCount;

//! The flock simulation and glyph operators
flock_sim* simulation;
glyph<thrust::device_vector<float3>::iterator, thrust::device_vector<float3>::iterator, thrust::device_vector<float>::iterator,
      thrust::device_vector<float3>::iterator, thrust::device_vector<float3>::iterator, thrust::device_vector<uint3>::iterator >* glyphs;

//! Initial positions and velocities for the boids
thrust::host_vector<float3>   inputPositionsHost;
thrust::device_vector<float3> inputPositions;
thrust::device_vector<float3> inputVelocities;

//! Vertices, normals, colors, vertex indices, and scalars for output
thrust::host_vector<float3>  vertices;
thrust::host_vector<float3>  normals;
thrust::host_vector<float4>  colors; 
thrust::host_vector<uint3>   indices;
thrust::device_vector<float> scalars;

//! Vertices, normals, and vertex indices for the sphere and arrow glyphs
thrust::device_vector<float3>  sphereGlyphVertices;
thrust::device_vector<float3>  sphereGlyphNormals;
thrust::device_vector<uint3>   sphereGlyphIndices;
thrust::device_vector<float3>  arrowGlyphVertices;
thrust::device_vector<float3>  arrowGlyphNormals;
thrust::device_vector<uint3>   arrowGlyphIndices;

//! VTK filters to produce the arrow and sphere glyphs
vtkArrowSource *arrowSource;
vtkSphereSource *sphereSource;
vtkPolyData *spherePoly;
vtkPolyData *arrowPoly;
vtkTriangleFilter *triangleFilter;
vtkPolyDataNormals *normalGenerator;

//! Camera and UI variables
int glyphMode;
bool simPaused;
float3 cameraPos;
float cameraFOV;
int gridSize;

//! Vertex buffer objects used by CUDA interop
#ifdef USE_INTEROP
  GLuint vboBuffers[4];  struct hipGraphicsResource* vboResources[4];
#endif


//==========================================================================
/*! 
    struct randomInit

    Initialize the vector elements with random values between the min and max
*/
//==========================================================================
struct randomInit : public thrust::unary_function<float3, float3>
{
    float minValue, maxValue;

    __host__ __device__
    randomInit(float minValue, float maxValue) : minValue(minValue), maxValue(maxValue) { };

    __host__ __device__
    float3 operator() (float3 i)
    {
      float3 result;
      result.x = minValue + (maxValue-minValue)*((rand() % 100000)/100000.0);
      result.y = minValue + (maxValue-minValue)*((rand() % 100000)/100000.0);
      result.z = minValue + (maxValue-minValue)*((rand() % 100000)/100000.0);
      return result;
    }
};


//==========================================================================
/*! 
    Extract vertices, normals, and vertex indices from a vtkPolyData instance

    \fn	copyPolyData
*/
//==========================================================================
void copyPolyData(vtkPolyData *polyData, thrust::device_vector<float3> &points, thrust::device_vector<float3> &vectors, thrust::device_vector<uint3> &indexes)
{
    // Extract the vertices and normals and copy to the output vectors
    vtkPoints* pts = polyData->GetPoints();
    vtkFloatArray* verts = vtkFloatArray::SafeDownCast(pts->GetData());
    vtkFloatArray* norms = vtkFloatArray::SafeDownCast(polyData->GetPointData()->GetNormals());
    float3* vData = (float3*)verts->GetPointer(0);
    float3* nData = (float3*)norms->GetPointer(0);
    points.assign(vData, vData+verts->GetNumberOfTuples());
    vectors.assign(nData, nData+norms->GetNumberOfTuples());

    // Extract the vertex indices from the cells and copy to the output vectors
    vtkCellArray* cellArray = polyData->GetPolys();
    vtkIdTypeArray* conn = cellArray->GetData();
    vtkIdType* cData = conn->GetPointer(0);
    for (int i=0; i<3*polyData->GetNumberOfPolys(); i++) cData[i] = cData[(i/3)*4+(i%3)+1];
    thrust::host_vector<uint> indexTemp;
    indexTemp.assign(cData, cData+3*polyData->GetNumberOfPolys());
    uint3* c3Data = (uint3*)(thrust::raw_pointer_cast(&*indexTemp.begin()));
    indexes.assign(c3Data, c3Data+polyData->GetNumberOfPolys());
}


//==========================================================================
/*! 
    Constructor for GLWindow class

    \fn	GLWindow::GLWindow
*/
//==========================================================================
GLWindow::GLWindow(QWidget *parent)
    : QGLWidget(QGLFormat(QGL::SampleBuffers), parent)
{
    // Start the QT callback timer
    setFocusPolicy(Qt::StrongFocus);
    timer = new QTimer(this);
    connect(timer, SIGNAL(timeout()), this, SLOT(updateGL()));
    timer->start(1);
}


//==========================================================================
/*! 
    Create the flock simulation and glyph operators

    \fn	GLWindow::initializeGL
*/
//==========================================================================
void GLWindow::initializeGL()
{
    // Initialize camera and UI variables
    qrot.set(0,0,0,1);
    frameCount = 0;
    gridSize = 256;
    glyphMode = 0;
    simPaused = false;
    cameraPos = make_float3(0.0f, 0.0f, 1.5*gridSize);
    cameraFOV = 60.0;

    // Set up basic OpenGL state and lighting
    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);
    float white[] = { 0.8, 0.8, 0.8, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { 0.0, 0.0, gridSize*1.5, 1.0 };
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);
    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
    glEnable(GL_COLOR_MATERIAL);

    // Initialize CUDA interop if it is being used
    #ifdef USE_INTEROP
      glewInit();
      cudaGLSetGLDevice(0);
    #endif

    // Initialize boid positions to random values and boid velocities to zero
    inputPositions.resize(INPUT_SIZE); inputPositionsHost.resize(INPUT_SIZE); 
    thrust::transform(inputPositionsHost.begin(), inputPositionsHost.end(), inputPositionsHost.begin(), randomInit(0.0f, 1.0f*gridSize));
    inputPositions = inputPositionsHost;
    thrust::fill(inputVelocities.begin(), inputVelocities.end(), make_float3(0.0f, 0.0f, 0.0f));

    // Set the boundaries of the simulation space
    float3 boundaryMin, boundaryMax;  
    boundaryMin.x = boundaryMin.y = boundaryMin.z = 0.0f;
    boundaryMax.x = boundaryMax.y = boundaryMax.z = gridSize;

    // Create the flock simulation instance
    simulation = new flock_sim(inputPositions, inputVelocities, boundaryMin, boundaryMax, 1.0f, 5.0f, 1.0f, 0.01f, 1.0025f, 30.0f, 5.0f, 30.0f, 4.0f, 10.0f, 0.01f, 0.5f);

    // Use VTK to generate a sphere glyph
    sphereSource = vtkSphereSource::New();
    sphereSource->SetThetaResolution(5);
    sphereSource->SetPhiResolution(5);
    sphereSource->Update();
    spherePoly = vtkPolyData::New();
    spherePoly->ShallowCopy(sphereSource->GetOutput());
    copyPolyData(spherePoly, sphereGlyphVertices, sphereGlyphNormals, sphereGlyphIndices);

    // Use VTK to generate an arrow glyph and its normals
    arrowSource = vtkArrowSource::New();
    arrowSource->Update();
    triangleFilter = vtkTriangleFilter::New();
    triangleFilter->SetInputConnection(arrowSource->GetOutputPort());
    triangleFilter->Update();
    arrowPoly = vtkPolyData::New();
    arrowPoly->ShallowCopy(triangleFilter->GetOutput());
    normalGenerator = vtkPolyDataNormals::New();
    normalGenerator->SetInput(arrowPoly);
    normalGenerator->ComputePointNormalsOn();
    normalGenerator->ComputeCellNormalsOff();
    normalGenerator->Update();
    arrowPoly = normalGenerator->GetOutput();
    copyPolyData(arrowPoly, arrowGlyphVertices, arrowGlyphNormals, arrowGlyphIndices);

    // Initialize glyph input scalars to the minimum simulation scalar value
    scalars.resize(INPUT_SIZE); 
    thrust::fill(scalars.begin(), scalars.end(), simulation->get_scalar_min());

    // Create the glyph operator instance
    glyphs = new glyph<thrust::device_vector<float3>::iterator, thrust::device_vector<float3>::iterator, thrust::device_vector<float>::iterator,
                       thrust::device_vector<float3>::iterator, thrust::device_vector<float3>::iterator, thrust::device_vector<uint3>::iterator>();
    
    // If using interop, initialize vertex buffer objects
    #ifdef USE_INTEROP
      int numPoints = INPUT_SIZE*std::max(sphereGlyphVertices.size(), arrowGlyphVertices.size());
      glGenBuffers(4, vboBuffers);
      for (int i=0; i<3; i++)
      {
        unsigned int bufferSize = (i == 1) ? numPoints*sizeof(float4) : numPoints*sizeof(float3);
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[i]);
        glBufferData(GL_ARRAY_BUFFER, bufferSize, 0, GL_DYNAMIC_DRAW);
      }
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[3]);
      glBufferData(GL_ARRAY_BUFFER, numPoints*sizeof(uint3), 0, GL_DYNAMIC_DRAW);
      glBindBuffer(GL_ARRAY_BUFFER, 0);
      for (int i=0; i<4; i++)
      {
        hipGraphicsGLRegisterBuffer(&(vboResources[i]), vboBuffers[i], cudaGraphicsMapFlagsWriteDiscard);
        glyphs->vboResources[i] = vboResources[i];
      }
    #endif

    // Enable OpenGL state for vertex, normal, and color arrays
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
}


//==========================================================================
/*! 
    Update the simulation and graphics

    \fn	GLWindow::paintGL
*/
//==========================================================================
void GLWindow::paintGL()
{
    // Stop the QT callback timer
    timer->stop();

    // Start timing this interval
    if (frameCount == 0) gettimeofday(&begin, 0);

    // Set up the OpenGL state
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    // Set up the projection and modelview matrices for the view
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(cameraFOV, 1.0f, 1.0f, gridSize*4.0f);
    glMatrixMode(GL_MODELVIEW);
    glPushMatrix();
    glLoadIdentity();
    gluLookAt(cameraPos.x, cameraPos.y, cameraPos.z, cameraPos.x, cameraPos.y, 0.0f, 0.0f, 1.0f, 0.0f);

    // Set up the current rotation and translation
    qrot.getRotMat(rotationMatrix);
    glMultMatrixf(rotationMatrix);
    glTranslatef(-(gridSize-1)/2, -(gridSize-1)/2, -(gridSize-1)/2);

    // If the simulation is not paused, compute the next simulation step, and apply the glyph operator to the result,
    // using either the arrow or sphere glyph
    int curGlyphMode = glyphMode;
    if (!simPaused)
    {
      (*simulation)();
      if (curGlyphMode == 0)
        (*glyphs)(simulation->positions_begin(), simulation->velocities_begin(), simulation->speeds_begin(), 
                  arrowGlyphVertices.begin(), arrowGlyphNormals.begin(), arrowGlyphIndices.begin(),     
                  INPUT_SIZE, arrowGlyphVertices.size(), arrowGlyphIndices.size(), 
                  simulation->get_scalar_min(), simulation->get_scalar_max());
      else 
        (*glyphs)(simulation->positions_begin(), simulation->velocities_begin(), scalars.begin(),
                  sphereGlyphVertices.begin(), sphereGlyphNormals.begin(), sphereGlyphIndices.begin(),     
                  INPUT_SIZE, sphereGlyphVertices.size(), sphereGlyphIndices.size(),
                  simulation->get_scalar_min(), simulation->get_scalar_max());
    }

    // If using interop, render the vertex buffer objects; otherwise, render the arrays
    #ifdef USE_INTEROP
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[0]);
      glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboBuffers[3]);
      glVertexPointer(3, GL_FLOAT, 0, 0);
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[1]);
      glColorPointer(4, GL_FLOAT, 0, 0);
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[2]);
      glNormalPointer(GL_FLOAT, 0, 0);
      int numIndices = INPUT_SIZE;
      if (curGlyphMode == 0) numIndices *= arrowGlyphIndices.size();
      else numIndices *= sphereGlyphIndices.size();
      glDrawElements(GL_TRIANGLES, 3*numIndices, GL_UNSIGNED_INT, 0);
      glBindBuffer(GL_ARRAY_BUFFER, 0);
      glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
    #else
      normals.assign(glyphs->normals_begin(), glyphs->normals_end());
      indices.assign(glyphs->indices_begin(), glyphs->indices_end());
      vertices.assign(glyphs->vertices_begin(), glyphs->vertices_end());
      colors.assign(thrust::make_transform_iterator(glyphs->scalars_begin(), color_map<float>(simulation->get_scalar_min(), simulation->get_scalar_max())),
    	            thrust::make_transform_iterator(glyphs->scalars_end(), color_map<float>(simulation->get_scalar_min(), simulation->get_scalar_max())));
      glNormalPointer(GL_FLOAT, 0, &normals[0]);
      glColorPointer(4, GL_FLOAT, 0, &colors[0]);
      glVertexPointer(3, GL_FLOAT, 0, &vertices[0]);
      glDrawElements(GL_TRIANGLES, 3*indices.size(), GL_UNSIGNED_INT, &indices[0]);
    #endif

    // Pop this OpenGL view matrix
    glPopMatrix();

    // Periodically output the framerate
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    frameCount++;
    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    if (seconds > 0.5f)
    {
      char title[256];
      sprintf(title, "Flock simulation, fps: %2.2f", float(frameCount)/seconds);
      std::cout << title << std::endl;
      seconds = 0.0f;
      frameCount = 0;
    }

    // Restart the QT callback timer
    timer->start(1);
}


//==========================================================================
/*! 
    Handle window resize event

    \fn	GLWindow::resizeGL
*/
//==========================================================================
void GLWindow::resizeGL(int width, int height)
{
    glViewport(0, 0, width, height);
}


//==========================================================================
/*! 
    Handle mouse press event

    \fn	GLWindow::mousePressEvent
*/
//==========================================================================
void GLWindow::mousePressEvent(QMouseEvent *event)
{
    lastPos = event->pos();
}


//==========================================================================
/*! 
    Handle mouse move event to rotate, translate, or zoom

    \fn	GLWindow::mouseMoveEvent
*/
//==========================================================================
void GLWindow::mouseMoveEvent(QMouseEvent *event)
{
    int dx = event->x() - lastPos.x();
    int dy = event->y() - lastPos.y();

    // Rotate, zoom, or translate the view
    if (event->buttons() & Qt::LeftButton)
    {
      Quaternion newRotX;
      newRotX.setEulerAngles(-0.2f*dx*3.14159f/180.0f, 0.0f, 0.0f);
      qrot.mul(newRotX);

      Quaternion newRotY;
      newRotY.setEulerAngles(0.0f, 0.0f, -0.2f*dy*3.14159f/180.0f);
      qrot.mul(newRotY);
    }
    else if (event->buttons() & Qt::RightButton)
    {
      cameraFOV += dy/20.0f;
    }
    else if (event->buttons() & Qt::MiddleButton)
    {
      cameraPos.x -= dx/2.0f;
      cameraPos.y += dy/2.0f; 
    }
    lastPos = event->pos();
}


//==========================================================================
/*! 
    Handle keyboard input event

    \fn	GLWindow::keyPressEvent
*/
//==========================================================================
void GLWindow::keyPressEvent(QKeyEvent *event)
{
   // Toggle the glyph type (spheres or arrows)
   if ((event->key() == 'g') || (event->key() == 'G'))
     if (!simPaused) glyphMode = 1 - glyphMode;

   // Pause or resume the simulation
   if ((event->key() == 'p') || (event->key() == 'P'))
     simPaused = !simPaused;
}


