#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <sstream>
#include <float.h>

#include "isorender.h"

#define PACKED __attribute__((packed))

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)


struct Rect
{
    int left,top,right,bottom;
};

struct TGAHeader
{
    unsigned char  identsize		;   // size of ID field that follows 18 uint8 header (0 usually)
    unsigned char  colourmaptype	;   // type of colour map 0=none, 1=has palette
    unsigned char  imagetype		;   // type of image 0=none,1=indexed,2=rgb,3=grey,+8=rle packed

    unsigned short colourmapstart	PACKED;   // first colour map entry in palette
    unsigned short colourmaplength	PACKED;   // number of colours in palette
    unsigned char  colourmapbits	;         // number of bits per palette entry 15,16,24,32

    unsigned short xstart		PACKED;   // image x origin
    unsigned short ystart		PACKED;   // image y origin
    unsigned short width		PACKED;   // image width in pixels
    unsigned short height		PACKED;   // image height in pixels
    unsigned char  bits			;         // image bits per pixel 8,16,24,32
    unsigned char  descriptor		;         // image descriptor bits (vh flip bits)

    inline bool IsFlippedHorizontal() const
    {
      return (descriptor & 0x10) != 0;
    }

    inline bool IsFlippedVertical() const
    {
      return (descriptor & 0x20) != 0;
    }
};


IsoRender::IsoRender()
{
    userMode = DEFAULT_MODE;
    animate = false;
    mouse_buttons = 0;
    translate = make_float3(0.0, 0.0, 0.0);
    frameCount = 0;
    lastIsovalue = -9999.9;
    lastPlaneLevel = -9999.9;
    lastThreshold = -9999.9;
    planeLevel = 0.0;
    includePlane = false;
    contours.resize(numDataSets);
    planeContours.resize(numDataSets);
    thresholds.resize(numDataSets);
    constantContours.resize(numDataSets);
    images.resize(numDataSets);
    planeFields.resize(numDataSets);
    polyData.resize(numDataSets);
    readers.resize(numDataSets);
    plyReaders.resize(numDataSets);
    timerTotal = 0; timerCount = 0;
    curFPS = 0.0f;
    for (int i=0; i<numDataSets; i++)
    {
      contours[i] = 0;
      planeContours[i] = 0;
      thresholds[i] = 0;
      constantContours[i] = 0;
      readers[i] = vtkXMLImageDataReader::New();
      polyData[i] = 0;
      plyReaders[i] = vtkPLYReader::New();
    }
}


void IsoRender::setIsovaluePct(float pct)
{
    isovaluePct = pct;
    isovalue = minIso + pct*(maxIso-minIso);

    thresholdPct = pct;
    threshold = minThreshold + pct*(maxThreshold-minThreshold);

    if (userMode == CUT_SURFACE_MODE) setPlaneLevelPct(pct);
}


void IsoRender::setPlaneLevelPct(float pct)
{
    planeLevelPct = pct;
    planeLevel = (-0.95 + (1.0-pct)*1.9)*(planeMax/2.0);
}


void IsoRender::setZoomLevelPct(float pct)
{
    if (pct > 1.0) pct = 1.0;  if (pct < 0.0) pct = 0.0;
    zoomLevelPct = pct;
    cameraFOV = 0.0 + zoomLevelBase*pct;
}


void IsoRender::resetView()
{
    qrot.set(qDefault.x, qDefault.y, qDefault.z, qDefault.w);
    zoomLevelPct = zoomLevelPctDefault;
    cameraFOV = 0.0 + zoomLevelBase*zoomLevelPct;
}


struct timeval begin, end, diff;
float seconds;
void IsoRender::display()
{
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    curFPS = 1.0/seconds;
    //std::cout << "Total fps: " << curFPS << std::endl;
    timerCount++;
    if (timerCount > 50)
    {
      timerTotal += 1.0/seconds;
      //std::cout << "Averages: " << timerCount << ": " << timerTotal/(1.0*timerCount-50.0) << std::endl;
    }
    gettimeofday(&begin, 0);
    //animate = true;
    if ((includeContours) && ((fabs(isovalue - lastIsovalue) > 0.01) || (animate)))
    {
      //std::cout << "Generating isovalue " << isovalue << std::endl;
#ifdef USE_INTEROP
      if (useInterop)
      {
        for (int i=0; i<3; i++) contours[dataSetIndex]->vboResources[i] = vboResources[i];
        for (int i=0; i<3; i++) contours[dataSetIndex]->vboBuffers[i] = vboBuffers[i];
        contours[dataSetIndex]->minIso = minIso;  contours[dataSetIndex]->maxIso = maxIso;
      }
#endif
      float value = isovalue;
      if (animate) value += (rand() % 100)/100.0;
      contours[dataSetIndex]->set_isovalue(value);
      (*(contours[dataSetIndex]))();
      lastIsovalue = isovalue;

      if (!useInterop)
      {
        vertices.assign(contours[dataSetIndex]->vertices_begin(), contours[dataSetIndex]->vertices_end());
        normals.assign(contours[dataSetIndex]->normals_begin(), contours[dataSetIndex]->normals_end());
        colors.assign(thrust::make_transform_iterator(contours[dataSetIndex]->scalars_begin(), color_map<float>(minIso, maxIso)),
                      thrust::make_transform_iterator(contours[dataSetIndex]->scalars_end(), color_map<float>(minIso, maxIso)));
      }
    }

    if ((includePlane) && (fabs(planeLevel - lastPlaneLevel) > 0.01))
    {
#ifdef USE_INTEROP
      if (useInterop)
      {
        for (int i=0; i<3; i++) planeContours[dataSetIndex]->vboResources[i] = planeResources[i];
        for (int i=0; i<3; i++) planeContours[dataSetIndex]->vboBuffers[i] = planeBuffers[i];
        planeContours[dataSetIndex]->minIso = minIso;  planeContours[dataSetIndex]->maxIso = maxIso;  planeContours[dataSetIndex]->colorFlip = useThreshold;
      }
#endif
      planeContours[dataSetIndex]->set_isovalue(planeLevel);
      (*(planeContours[dataSetIndex]))();
      lastPlaneLevel = planeLevel;
      if (!useInterop)
      {
        planeVertices.assign(planeContours[dataSetIndex]->vertices_begin(), planeContours[dataSetIndex]->vertices_end());
        planeNormals.assign(planeContours[dataSetIndex]->normals_begin(), planeContours[dataSetIndex]->normals_end());
        planeColors.assign(thrust::make_transform_iterator(planeContours[dataSetIndex]->scalars_begin(), color_map<float>(minIso, maxIso, useThreshold)),
                           thrust::make_transform_iterator(planeContours[dataSetIndex]->scalars_end(), color_map<float>(minIso, maxIso, useThreshold)));
      }
    }

    if ((includeThreshold) && (fabs(threshold - lastThreshold) > 0.01))
    {
      //std::cout << "Generating threshold " << thresholdFloor << " " << threshold << std::endl;
#ifdef USE_INTEROP
      if (useInterop)
      {
        for (int i=0; i<3; i++) thresholds[dataSetIndex]->vboResources[i] = vboResources[i];
        for (int i=0; i<3; i++) thresholds[dataSetIndex]->vboBuffers[i] = vboBuffers[i];
        thresholds[dataSetIndex]->minThresholdRange = minThreshold;  thresholds[dataSetIndex]->maxThresholdRange = maxThreshold;
      }
#endif
      thresholds[dataSetIndex]->set_threshold_range(thresholdFloor, threshold);
      thresholds[dataSetIndex]->colorFlip = true;
      (*(thresholds[dataSetIndex]))();
      lastThreshold = threshold;
      if (!useInterop)
      {
        thrust::device_vector<float4> device_colors;
        vertices.resize(thresholds[dataSetIndex]->vertices_end() - thresholds[dataSetIndex]->vertices_begin());
        normals.resize(thresholds[dataSetIndex]->normals_end() - thresholds[dataSetIndex]->normals_begin());
        device_colors.resize(thresholds[dataSetIndex]->vertices_end() - thresholds[dataSetIndex]->vertices_begin());
        thrust::copy(thresholds[dataSetIndex]->normals_begin(), thresholds[dataSetIndex]->normals_end(), normals.begin());
        thrust::copy(thresholds[dataSetIndex]->vertices_begin(),
                     thresholds[dataSetIndex]->vertices_end(), vertices.begin());
        thrust::transform(thresholds[dataSetIndex]->scalars_begin(), thresholds[dataSetIndex]->scalars_end(),
                          device_colors.begin(), color_map<float>(minThreshold, maxThreshold, true));
        colors = device_colors;
      }
    }

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(cameraFOV, 2.0, zNear, zFar);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(center_pos.x, center_pos.y, cameraZ,
                  center_pos.x, center_pos.y, center_pos.z,
                  camera_up.x, camera_up.y, camera_up.z);
    glPushMatrix();

    float3 center = make_float3(center_pos.x, center_pos.y, center_pos.z);

    qrot.getRotMat(rotationMatrix);
    glMultMatrixf(rotationMatrix);

    GLfloat matrix[16];
    glGetFloatv(GL_MODELVIEW_MATRIX, matrix);
    float3 offset = make_float3(matrix[0]*center.x + matrix[1]*center.y + matrix[2]*center.z, matrix[4]*center.x + matrix[5]*center.y + matrix[6]*center.z,
                                matrix[8]*center.x + matrix[9]*center.y + matrix[10]*center.z);
    offset.x = center.x - offset.x; offset.y = center.y - offset.y; offset.z = center.z - offset.z;
    glTranslatef(-offset.x, -offset.y, -offset.z);

    if (includeContours)
    {
      glEnableClientState(GL_VERTEX_ARRAY);

      if (bigDemo)
      {
        glDisableClientState(GL_COLOR_ARRAY);
        color_map<float> isoColor(minIso, maxIso);
        float4 icolor = isoColor(isovalue);
        glColor3f(icolor.x, icolor.y, icolor.z);
      }
      else
      {
        glEnableClientState(GL_COLOR_ARRAY);
      }
      glEnableClientState(GL_NORMAL_ARRAY);

      if (useInterop)
      {
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[0]);
        glVertexPointer(4, GL_FLOAT, 0, 0);

        if (!bigDemo)
        {
          glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[1]);
          glColorPointer(4, GL_FLOAT, 0, 0);
        }

        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[2]);
        glNormalPointer(GL_FLOAT, 0, 0);

        glDrawArrays(GL_TRIANGLES, 0, contours[dataSetIndex]->num_total_vertices);
      }
      else
      {
        glNormalPointer(GL_FLOAT, 0, &normals[0]);
        glColorPointer(4, GL_FLOAT, 0, &colors[0]);
        glVertexPointer(4, GL_FLOAT, 0, &vertices[0]);
        glDrawArrays(GL_TRIANGLES, 0, vertices.size());
      }
    }

    if (includePlane)
    {
      glEnableClientState(GL_VERTEX_ARRAY);
      glEnableClientState(GL_COLOR_ARRAY);
      glEnableClientState(GL_NORMAL_ARRAY);

      if (useInterop)
      {
        glBindBuffer(GL_ARRAY_BUFFER, planeBuffers[0]);
        glVertexPointer(4, GL_FLOAT, 0, 0);

        glBindBuffer(GL_ARRAY_BUFFER, planeBuffers[1]);
        glColorPointer(4, GL_FLOAT, 0, 0);

        glBindBuffer(GL_ARRAY_BUFFER, planeBuffers[2]);
        glNormalPointer(GL_FLOAT, 0, 0);

        glDrawArrays(GL_TRIANGLES, 0, planeContours[dataSetIndex]->num_total_vertices);
      }
      else
      {
        glNormalPointer(GL_FLOAT, 0, &planeNormals[0]);
        glColorPointer(4, GL_FLOAT, 0, &planeColors[0]);
        glVertexPointer(4, GL_FLOAT, 0, &planeVertices[0]);
        glDrawArrays(GL_TRIANGLES, 0, planeVertices.size());
      }
    }

    if (includeThreshold)
    {
      glEnableClientState(GL_VERTEX_ARRAY);
      glEnableClientState(GL_COLOR_ARRAY);
      glEnableClientState(GL_NORMAL_ARRAY);
      if (useInterop)
      {
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[0]);
        glVertexPointer(4, GL_FLOAT, 0, 0);

        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[1]);
        glColorPointer(4, GL_FLOAT, 0, 0);

        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[2]);
        glNormalPointer(GL_FLOAT, 0, 0);

        glDrawArrays(GL_QUADS, 0, thresholds[dataSetIndex]->num_total_vertices);
      }
      else
      {
        glNormalPointer(GL_FLOAT, 0, &normals[0]);
        glColorPointer(4, GL_FLOAT, 0, &colors[0]);
        glVertexPointer(4, GL_FLOAT, 0, &vertices[0]);
        glDrawArrays(GL_QUADS, 0, vertices.size());
      }
    }

    if (includePolygons)
    {
      polyTriangles = polyData[dataSetIndex]->GetPolys();
      polyTriangles->InitTraversal();

      glDisableClientState(GL_VERTEX_ARRAY);
      glDisableClientState(GL_COLOR_ARRAY);
      glDisableClientState(GL_NORMAL_ARRAY);
      glColor3f(0.5, 0.5, 0.5);

      glPushMatrix();
      glTranslatef(xMax/2.0, yMax/2.0, 0.0);

      glBegin(GL_TRIANGLES);
      for (int i=0; i<polyTriangles->GetNumberOfCells(); i++)
      {
        polyTriangles->GetNextCell(npts, curTriangle);
    	for (int j=0; j<npts; j++)
    	{
    	  double p[3];
          polyData[dataSetIndex]->GetPoint(curTriangle[j], p);
          glVertex3f(polyScale*(p[0]-polyOffset.x), polyScale*(p[1]-polyOffset.y), polyScale*(p[2]-polyOffset.z));
    	}
      }
      glEnd(); gettimeofday(&begin, 0);
      glPopMatrix();
    }

    if (includeConstantContours)
    {
      glEnableClientState(GL_VERTEX_ARRAY);
      glDisableClientState(GL_COLOR_ARRAY);
      glColor3f(0.2, 0.2, 0.2);
      glEnableClientState(GL_NORMAL_ARRAY);

      if (useInterop)
      {
        glBindBuffer(GL_ARRAY_BUFFER, constantBuffers[0]);
        glVertexPointer(4, GL_FLOAT, 0, 0);

        glBindBuffer(GL_ARRAY_BUFFER, constantBuffers[2]);
        glNormalPointer(GL_FLOAT, 0, 0);

        glDrawArrays(GL_TRIANGLES, 0, numConstantVertices);
      }
      else
      {
        glNormalPointer(GL_FLOAT, 0, &constantNormals[0]);
        glVertexPointer(4, GL_FLOAT, 0, &constantVertices[0]);
        glDrawArrays(GL_TRIANGLES, 0, constantVertices.size());
      }
    }

    glPopMatrix();

    if (showLabels)
    {
      glDisable(GL_LIGHTING);
      glMatrixMode(GL_PROJECTION);
      glPushMatrix();
      glLoadIdentity();
      glOrtho(0, viewportWidth, 0, viewportHeight, -100000.0, 100000.0);
      glMatrixMode(GL_MODELVIEW);
      glPushMatrix();
      glLoadIdentity();
      glDisable(GL_DEPTH_TEST);
      glDepthMask(GL_FALSE);
      glColor3f(1.0, 1.0, 1.0);
      char line[256] = "";

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
      sprintf(line, "CUDA Backend");
#endif
#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_OMP
      sprintf(line, "OpenMP Backend");
#endif
      glRasterPos2f(10.0, viewportHeight/8.0);
      for (int c=0; c<strlen(line); c++)
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, line[c]);

/*#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
      sprintf(line, "Quadro 6000 GPU (448 cores)");
#endif
#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_OMP
      sprintf(line, "Intel Xeon 2.67 GHz CPU (12 cores)");
#endif
      glRasterPos2f(10.0, viewportHeight/8.0-20);
      for (int c=0; c<strlen(line); c++)
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, line[c]);*/

      sprintf(line, "Dimensions: %d x %d x %d", xMax-xMin+1, yMax-yMin+1, zMax-zMin+1);
      glRasterPos2f(10.0, viewportHeight/8.0-40);
      for (int c=0; c<strlen(line); c++)
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, line[c]);

      sprintf(line, "Points: %d", (xMax-xMin+1)*(yMax-yMin+1)*(zMax-zMin+1));
      glRasterPos2f(10.0, viewportHeight/8.0-60);
      for (int c=0; c<strlen(line); c++)
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, line[c]);

      sprintf(line, "FPS: %.1f", curFPS);
      glRasterPos2f(10.0, viewportHeight/8.0-80);
      for (int c=0; c<strlen(line); c++)
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, line[c]);

      sprintf(line, "%.1f", maxIso);
      glRasterPos2f(20, 400);
      for (int c=0; c<strlen(line); c++)
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, line[c]);

      sprintf(line, "%.1f", minIso);
      glRasterPos2f(20, 200);
      for (int c=0; c<strlen(line); c++)
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, line[c]);

      sprintf(line, "%.1f", (maxIso+minIso)/2.0);
      glRasterPos2f(20, 300);
      for (int c=0; c<strlen(line); c++)
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, line[c]);

      color_map<float> colorMap(minIso, maxIso, useThreshold);
      float h = 10; float w = 30;
      float x = 100.0;  float y = 200.0;
      glBegin(GL_TRIANGLES);
        for (int i=0; i<20; i++)
        {
    	  float4 mapColor = colorMap(minIso + (i/19.0)*(maxIso-minIso));
    	  glColor4f(mapColor.x, mapColor.y, mapColor.z, mapColor.w);
          glVertex3f(x, y, 0);
          glVertex3f(x+w, y, 0);
          glVertex3f(x, y+h, 0);
          glVertex3f(x+w, y, 0);
          glVertex3f(x+w, y+h, 0);
          glVertex3f(x, y+h, 0);
          y += h;
        }
      glEnd();

      glEnable(GL_LIGHTING);
      glDepthMask(GL_TRUE);
      glEnable(GL_DEPTH_TEST);
      glMatrixMode(GL_PROJECTION);
      glPopMatrix();
      glMatrixMode(GL_MODELVIEW);
      glPopMatrix();
    }

    if (saveFrames)
    {
      Rect glRect;
      glGetIntegerv( GL_VIEWPORT, (int*)&glRect );
      char screenShotFile[256];
      if (frameCount < 10)
        sprintf(screenShotFile, "%s/Frame00%d.tga", frameDirectory, frameCount);
      else if (frameCount < 100)
        sprintf(screenShotFile, "%s/Frame0%d.tga", frameDirectory, frameCount);
      else 
        sprintf(screenShotFile, "%s/Frame%d.tga", frameDirectory, frameCount);
      screenShot(screenShotFile,glRect.right - glRect.left,glRect.bottom - glRect.top,false);
      std::cout << "Output frame " << frameCount << std::endl;
      frameCount++;
    }
}


void IsoRender::cleanup()
{
#ifdef USE_INTEROP
    if (useInterop)
    {
      printf("Deleting VBO\n");
      if (vboBuffers[0])
      {
        for (int i=0; i<3; i++) hipGraphicsUnregisterResource(vboResources[i]);
        for (int i=0; i<3; i++)
        {
          glBindBuffer(1, vboBuffers[i]);
      	  glDeleteBuffers(1, &(vboBuffers[i]));
       	  vboBuffers[i] = 0;
        }
      }
      if (planeBuffers[0])
      {
        for (int i=0; i<3; i++) hipGraphicsUnregisterResource(planeResources[i]);
        for (int i=0; i<3; i++)
        {
          glBindBuffer(1, planeBuffers[i]);
    	  glDeleteBuffers(1, &(planeBuffers[i]));
    	  planeBuffers[i] = 0;
        }
      }
      if (constantBuffers[0])
      {
        for (int i=0; i<3; i++) if (i != 1) hipGraphicsUnregisterResource(constantResources[i]);
        for (int i=0; i<3; i++)
        {
          if (i == 1) continue;
          glBindBuffer(1, constantBuffers[i]);
    	  glDeleteBuffers(1, &(constantBuffers[i]));
    	  constantBuffers[i] = 0;
        }
      }
    }
    else
#endif
    {
      vertices.clear(); normals.clear(); colors.clear();
      planeVertices.clear(); planeNormals.clear(); planeColors.clear();
      constantVertices.clear(); constantNormals.clear();
    }

    for (int i=0; i<numDataSets; i++)
    {
      if (contours[i]) contours[i]->freeMemory();
      if (thresholds[i]) thresholds[i]->freeMemory();
      if (planeContours[i]) planeContours[i]->freeMemory();
      if (constantContours[i]) constantContours[i]->freeMemory();
    }
}


void IsoRender::initGL(bool aAllowInterop, bool aBigDemo, bool aShowLabels, int aDataSet)
{
    showLabels = aShowLabels;
    if (showLabels)
    {
      int argc = 0; char **argv = 0; glutInit(&argc, argv);
    }

#ifdef USE_INTEROP
    useInterop = aAllowInterop;
#else
    useInterop = false;
#endif
    bigDemo = aBigDemo;

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    float white[] = { 0.5, 0.5, 0.5, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { 100.0, 100.0, -100.0, 1.0 };
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

    glMatrixMode(GL_PROJECTION);
    gluPerspective(cameraFOV, 2.0, zNear, zFar);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(center_pos.x, center_pos.y, cameraZ,
              center_pos.x, center_pos.y, center_pos.z,
              camera_up.x, camera_up.y, camera_up.z);

#ifdef USE_INTEROP
    if (useInterop)
    {
      glewInit();
      cudaGLSetGLDevice(0);

      createBuffers();
    }
#endif

    //printf("Error code: %s\n", hipGetErrorString(errorCode));
    if (userMode == DEFAULT_MODE) read(aDataSet);
    else read(userFileName, userMode);
}


void IsoRender::timeContours()
{
    contours[dataSetIndex]->useInterop = false;
    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);
    for (int i=0; i<numIters; i++)
    {
      isovalue = minIso; // + ((1.0*i)/(1.0*numIters))*(maxIso - minIso);
      //std::cout << "Generating isovalue " << isovalue << std::endl;
      contours[dataSetIndex]->set_isovalue(isovalue);
      (*(contours[dataSetIndex]))();
    }
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    std::cout << "contour fps: " << numIters/seconds << std::endl;
}


void IsoRender::screenShot(std::string fileName, unsigned int width, unsigned int height, bool includeAlpha)
{
    std::cout << "Screen shot" << std::endl;
    unsigned int pixelSize = 3;
    unsigned int pixelSizeBits = 24;
    GLenum pixelFormat = GL_BGR_EXT;

    if (includeAlpha)
    {
      pixelSize = sizeof(unsigned int);
      pixelSizeBits = 32;
      pixelFormat = GL_BGRA_EXT;
    }

    char* pBuffer = new char[pixelSize*width*height ];

    glReadPixels( 0,0,width,height,pixelFormat,GL_UNSIGNED_BYTE,pBuffer );

    TGAHeader tgah;
    memset( &tgah,0,sizeof(TGAHeader) );

    tgah.bits = pixelSizeBits;
    tgah.height = height;
    tgah.width = width;
    tgah.imagetype = 2;

    std::ofstream ofile( fileName.c_str(), std::ios_base::binary );

    ofile.write( (char*)&tgah, sizeof(tgah) );
    ofile.write( pBuffer, pixelSize*width*height );

    ofile.close();

    delete [] pBuffer;
}


void IsoRender::createBuffers()
{
#ifdef USE_INTEROP
    // initialize contour buffer objects
    glGenBuffers(3, vboBuffers);
    for (int i=0; i<3; i++)
    {
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[i]);
      glBufferData(GL_ARRAY_BUFFER, BUFFER_SIZE*sizeof(float4), 0, GL_DYNAMIC_DRAW);
    }
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    for (int i=0; i<3; i++) hipGraphicsGLRegisterBuffer(&(vboResources[i]), vboBuffers[i], cudaGraphicsMapFlagsWriteDiscard);

    // initialize plane buffer objects
    glGenBuffers(3, planeBuffers);
    for (int i=0; i<3; i++)
    {
      glBindBuffer(GL_ARRAY_BUFFER, planeBuffers[i]);
      glBufferData(GL_ARRAY_BUFFER, BUFFER_SIZE*sizeof(float4), 0, GL_DYNAMIC_DRAW);
    }
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    for (int i=0; i<3; i++) hipGraphicsGLRegisterBuffer(&(planeResources[i]), planeBuffers[i], cudaGraphicsMapFlagsWriteDiscard);

    // initialize constant contour buffer objects
    glGenBuffers(3, constantBuffers);
    for (int i=0; i<3; i++)
    {
      if (i == 1) continue;
      glBindBuffer(GL_ARRAY_BUFFER, constantBuffers[i]);
      glBufferData(GL_ARRAY_BUFFER, BUFFER_SIZE*sizeof(float3), 0, GL_DYNAMIC_DRAW);
    }
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    for (int i=0; i<3; i++) if (i != 1) hipGraphicsGLRegisterBuffer(&(constantResources[i]), constantBuffers[i], cudaGraphicsMapFlagsWriteDiscard);
#endif
}


void IsoRender::createOperators()
{
    if ((contours[dataSetIndex] == 0) && (thresholds[dataSetIndex] == 0))
    {
      readers[dataSetIndex]->Update();
      output = readers[dataSetIndex]->GetOutput();
      images[dataSetIndex] = new vtk_image3d<SPACE>(output);
    }

    if ((includeContours) && (contours[dataSetIndex] == 0))
    {
      contours[dataSetIndex] = new marching_cube<vtk_image3d<SPACE>, vtk_image3d<SPACE> >(*(images[dataSetIndex]), *(images[dataSetIndex]), isovalue);
      contours[dataSetIndex]->useInterop = useInterop;
      contours[dataSetIndex]->discardMinVals = discardMinVals;
    }

    if ((includePlane) && (planeContours[dataSetIndex] == 0))
    {
      planeFields[dataSetIndex] = new plane_field<SPACE>(make_float3((xMax-xMin+1)/2.0, (yMax-yMin+1)/2.0, (zMax-zMin+1)/2.0), plane_normal, xMax-xMin+1, yMax-yMin+1, zMax-zMin+1);
      planeContours[dataSetIndex] = new marching_cube<plane_field<SPACE>, vtk_image3d<SPACE> >(*(planeFields[dataSetIndex]), *(images[dataSetIndex]), isovalue);
      planeContours[dataSetIndex]->useInterop = useInterop;
    }

    if (includeThreshold && (thresholds[dataSetIndex] == 0))
    {
      thresholds[dataSetIndex] = new threshold_geometry<vtk_image3d<SPACE> >(*(images[dataSetIndex]), thresholdFloor, threshold);
      thresholds[dataSetIndex]->useInterop = useInterop;
    }

    if ((includeConstantContours) && (constantContours[dataSetIndex] == 0))
    {
      if (useContours) constantContours[dataSetIndex] = contours[dataSetIndex];
      else constantContours[dataSetIndex] = new marching_cube<vtk_image3d<SPACE>, vtk_image3d<SPACE> >(*(images[dataSetIndex]), *(images[dataSetIndex]), isovalue);
    }

    if (includeConstantContours)
    {
#ifdef USE_INTEROP
      if (useInterop)
      {
        for (int i=0; i<3; i++) if (i != 1) constantContours[dataSetIndex]->vboResources[i] = constantResources[i];
        for (int i=0; i<3; i++) if (i != 1) constantContours[dataSetIndex]->vboBuffers[i] = constantBuffers[i];
	constantContours[dataSetIndex]->vboResources[1] = 0;
	constantContours[dataSetIndex]->vboSize = 0;
	constantContours[dataSetIndex]->useInterop = useInterop;
      }
#endif
      constantContours[dataSetIndex]->discardMinVals = false;
      constantContours[dataSetIndex]->set_isovalue(-99999.9);
      constantContours[dataSetIndex]->discardMinVals = false;
      (*(constantContours[dataSetIndex]))();
      if (!useInterop)
      {
        constantVertices.assign(constantContours[dataSetIndex]->vertices_begin(), constantContours[dataSetIndex]->vertices_end());
	    constantNormals.assign(constantContours[dataSetIndex]->normals_begin(), constantContours[dataSetIndex]->normals_end());
      }
      numConstantVertices = constantContours[dataSetIndex]->num_total_vertices;
      if (!useContours) constantContours[dataSetIndex]->freeMemory(false);
      constantContours[dataSetIndex]->discardMinVals = true;
    }

#ifdef USE_INTEROP
    if (includeContours) contours[dataSetIndex]->vboSize = BUFFER_SIZE;
    if (includePlane) planeContours[dataSetIndex]->vboSize = BUFFER_SIZE;
    if (includeThreshold) thresholds[dataSetIndex]->vboSize = BUFFER_SIZE;
#endif

}


int IsoRender::read(char* aFileName, int aMode)
{
    dataSetIndex = 1;  numIters = 0;  saveFrames = 0;

    char filename[1024];
    sprintf(filename, "%s/%s", STRINGIZE_VALUE_OF(DATA_DIRECTORY), userFileName);

    int fileFound = readers[dataSetIndex]->CanReadFile(filename);
    if (fileFound == 0) sprintf(filename, userFileName);
    readers[dataSetIndex]->SetFileName(filename);

    readers[dataSetIndex]->Update();
    output = readers[dataSetIndex]->GetOutput();

    int dims[3];  output->GetDimensions(dims);
    xMin = yMin = zMin = 0;  xMax = dims[0]-1;  yMax = dims[1]-1;  zMax = dims[2]-1;

    NPoints = (xMax - xMin + 1) * (yMax - yMin + 1) * (zMax - zMin + 1);
    center_pos = make_float3((xMax - xMin + 1)/2.0, (yMax - yMin + 1)/2.0, (zMax - zMin + 1)/2.0);

    if (userRange)
    {
      minIso = userMin;  maxIso = userMax;  minThreshold = thresholdFloor = userMin;  maxThreshold = userMax;
    }
    else
    {
      float* rawData = (float*)(output->GetScalarPointer());
      float minVal = FLT_MAX;  float maxVal = FLT_MIN;
      for (int i=0; i<NPoints; i++)
      {
        if (rawData[i] < minVal) minVal = rawData[i];
	if (rawData[i] > maxVal) maxVal = rawData[i];
      }
      minIso = minVal + (0.01*(maxVal - minVal));  maxIso = maxVal - (0.01*(maxVal - minVal));
      minThreshold = thresholdFloor = minIso;  maxThreshold = maxIso;
    }
    isovaluePct = 0.5;

    discardMinVals = true;
    plane_normal.x = 0.0;  plane_normal.y = 0.0;  plane_normal.z = 1.0;  planeLevelPct = 0.5;
    zoomLevelPctDefault = 0.5;  cameraFOV = 36.0;  cameraZ = 3.0*std::max(std::max(xMax, yMax), zMax);
    zFar = 2.0*cameraZ;  zNear = zFar/10.0;
    planeMax = (xMax-xMin)*plane_normal.x + (yMax-yMin)*plane_normal.y + (zMax-zMin)*plane_normal.z;

    zoomLevelBase = cameraFOV;
    qDefault.set(0, 0, 0, 1);  qrot.set(qDefault.x, qDefault.y, qDefault.z, qDefault.w);
    zoomLevelPct = zoomLevelPctDefault;
    isovalue = minIso;  threshold = minThreshold;

    cameraFOV = zoomLevelBase*zoomLevelPct;  camera_up = make_float3(0,1,0);

    includeContours = (userMode == ISOSURFACE_MODE);  includeThreshold = (userMode == THRESHOLD_MODE);  includePlane = (userMode == CUT_SURFACE_MODE);
    includeConstantContours = false;  includePolygons = false;
    useContours = includeContours; useThreshold = includeThreshold; useConstantContours = includeConstantContours;

    createOperators();

    lastIsovalue = -9999.9;
    lastPlaneLevel = -9999.9;
    lastThreshold = -9999.9;
    std::cout << "Read user file " << filename << std::endl;

    return 0;
}


int IsoRender::read(int aDataSetIndex, int aNumIters, int aSaveFrames, char* aFrameDirectory)
{
    dataSetIndex = aDataSetIndex;
    numIters = aNumIters;
    saveFrames = aSaveFrames;
    if (saveFrames) strcpy(frameDirectory, aFrameDirectory);

    char metafile[1024]; char fname[1024]; char pname[1024]; char dtag[1024]; float qx, qy, qz, qw;
    fname[0] = 0; pname[0] = 0;
    sprintf(metafile, "%s/dataset%d.txt", STRINGIZE_VALUE_OF(DATA_DIRECTORY), dataSetIndex);

    std::string line, tag;
    std::ifstream myfile(metafile);
    if (myfile.is_open())
    {
      while (myfile.good())
      {
        getline (myfile,line);
        std::stringstream lineStream(line);
        lineStream >> tag;

        if (tag.compare("data") == 0)       sscanf(line.c_str(), "%s %s", dtag, fname);
        if (tag.compare("dimensions") == 0) sscanf(line.c_str(), "%s %d %d %d %d %d %d", dtag, &xMin, &xMax, &yMin, &yMax, &zMin, &zMax);
        if (tag.compare("polys") == 0)      sscanf(line.c_str(), "%s %s", dtag, pname);
        if (tag.compare("isovalues") == 0)  sscanf(line.c_str(), "%s %f %f %f", dtag, &minIso, &maxIso, &isovaluePct);
        if (tag.compare("thresholds") == 0) sscanf(line.c_str(), "%s %f %f %f", dtag, &minThreshold, &maxThreshold, &thresholdFloor);
        if (tag.compare("zoom") == 0)       sscanf(line.c_str(), "%s %f %f %f", dtag, &cameraFOV, &cameraZ, &zoomLevelPctDefault);
        if (tag.compare("quaternion") == 0) sscanf(line.c_str(), "%s %f %f %f %f", dtag, &qx, &qy, &qz, &qw);
        if (tag.compare("plane") == 0)      sscanf(line.c_str(), "%s %f %f %f %f", dtag, &(plane_normal.x), &(plane_normal.y), &(plane_normal.z), &planeLevelPct);
        if (tag.compare("include") == 0)    sscanf(line.c_str(), "%s %d %d %d %d", dtag, &includeContours, &includePlane, &includeThreshold, &includeConstantContours);
        if (tag.compare("discard") == 0)    sscanf(line.c_str(), "%s %d", dtag, &discardMinVals);
      }
      myfile.close();
    }
    else cout << "Unable to open file";

    char filename[1024];
    sprintf(filename, "%s/%s", STRINGIZE_VALUE_OF(DATA_DIRECTORY), fname);
    readers[dataSetIndex]->SetFileName(filename);
    NPoints = (xMax - xMin + 1) * (yMax - yMin + 1) * (zMax - zMin + 1);
    center_pos = make_float3((xMax - xMin + 1)/2.0, (yMax - yMin + 1)/2.0, (zMax - zMin + 1)/2.0);
    planeMax = (xMax-xMin)*plane_normal.x + (yMax-yMin)*plane_normal.y + (zMax-zMin)*plane_normal.z;

    zoomLevelBase = cameraFOV;  zNear = 200.0;  zFar = 4000.0;
    qDefault.set(qx, qy, qz, qw);  qrot.set(qDefault.x, qDefault.y, qDefault.z, qDefault.w);
    zoomLevelPct = zoomLevelPctDefault;
    isovalue = minIso;  threshold = minThreshold;

    cameraFOV = zoomLevelBase*zoomLevelPct;  camera_up = make_float3(0,1,0);
    useContours = includeContours; useThreshold = includeThreshold; useConstantContours = includeConstantContours;
    includePolygons = (pname[0] != 0);

    if (includePolygons)
    {
      sprintf(filename, "%s/%s", STRINGIZE_VALUE_OF(DATA_DIRECTORY), pname);
      plyReaders[dataSetIndex]->SetFileName(filename);
      if ((contours[dataSetIndex] == 0) && (thresholds[dataSetIndex] == 0))
      {
        plyReaders[dataSetIndex]->Update();
        polyData[dataSetIndex] = plyReaders[dataSetIndex]->GetOutput();

        polyAvgX = polyAvgY = polyAvgZ = 0.0;
        polyTriangles = polyData[dataSetIndex]->GetPolys();
        ncells = polyTriangles->GetNumberOfCells();
        for (int i=0; i<ncells; i++)
        {
          polyTriangles->GetNextCell(npts, curTriangle);
          for (int j=0; j<npts; j++)
          {
            double p[3];
            polyData[dataSetIndex]->GetPoint(curTriangle[j], p);
            polyAvgX += p[0]; polyAvgY += p[1]; polyAvgZ += p[2];
          }
        }
        polyOffset.x = polyAvgX/(3.0*ncells);
        polyOffset.y = polyAvgY/(3.0*ncells);
        polyOffset.z = 0.0;
        polyScale = 0.25;
      }
    }

    createOperators();

    lastIsovalue = -9999.9;
    lastPlaneLevel = -9999.9;
    lastThreshold = -9999.9;
    std::cout << "Read file " << dataSetIndex << std::endl;

    return 0;
}
