#include "hip/hip_runtime.h"
/*
 * simple_opengl.cu
 *
 *  Created on: Oct 7, 2011
 *      Author: ollie
 */
#ifdef __APPLE__
    #include <GL/glew.h>
    #include <OpenGL/OpenGL.h>
    #include <GLUT/glut.h>
#else
    #include <GL/glew.h>
    #include <GL/glut.h>
    #include <GL/gl.h>
#endif

#include <cuda_gl_interop.h>

#include <thrust/device_vector.h>
#include <thrust/distance.h>

#include <piston/implicit_function.h>
#include <piston/cutil_math.h>
#include <piston/image2d.h>

//#define SPACE  thrust::host_system_tag
#define SPACE thrust::device_system_tag

using namespace piston;
static const int GRID_SIZE = 4;

struct sine_wave : public piston::image2d<int, float4, SPACE>
{
    struct sine_functor : public piston::implicit_function2d<int, float4>
    {
	typedef piston::implicit_function2d<int, float4> Parent;
	typedef typename Parent::InputType InputType;

	int xdim;
	int ydim;
	float time;

	sine_functor(int xdim, int ydim, float time) :
	    xdim(xdim), ydim(ydim), time(time) {}

	__host__ __device__
	float4 operator()(InputType pos) const {
	    unsigned int x = thrust::get<0>(pos);
	    unsigned int y = thrust::get<1>(pos);

	    // calculate uv coordinates
	    float u = x / (float) xdim;
	    float v = y / (float) ydim;
	    u = u*2.0f - 1.0f;
	    v = v*2.0f - 1.0f;

	    // calculate simple sine wave pattern
	    float freq = 4.0f;

	    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	    // write output vertex
	    return make_float4(u, w, v, 1.0f);
	}
    };

    typedef piston::image2d<int, float4, SPACE> Parent;
    typedef thrust::transform_iterator<sine_functor,
				       typename Parent::GridCoordinatesIterator> PointDataIterator;
    float time;
    PointDataIterator point_data_iterator;

    sine_wave(int xdim, int ydim, float time = 0.0f) :
	Parent(xdim, ydim),
	time(time),
	point_data_iterator(this->grid_coordinates_iterator, sine_functor(xdim, ydim, time)){}

    void resize(int xdim, int ydim) {
	Parent::resize(xdim, ydim);
	point_data_iterator = thrust::make_transform_iterator(grid_coordinates_iterator,
	                                                      sine_functor(xdim, ydim, time));
    }

    void set_time(float time) {
	this->time = time;
	point_data_iterator = thrust::make_transform_iterator(grid_coordinates_iterator,
	                                                      sine_functor(xdim, ydim, time));
    }

    PointDataIterator point_data_begin() {
	return point_data_iterator;
    }

    PointDataIterator point_data_end() {
	return point_data_iterator + this->NPoints;
    }
};

struct print_tuple2 : public thrust::unary_function<thrust::tuple<int, int>, void>
{
    __host__ __device__
    void operator() (thrust::tuple<int, int> pos) {
	std::cout << "(" << thrust::get<0>(pos) << ", "
		         << thrust::get<1>(pos) << ")" << std::endl;
    }
};

struct print_float4 : public thrust::unary_function<float4, void>
{
	__host__ __device__
	void operator() (float4 p) {
	    std::cout << "(" << p.x << ", " << p.y << ", " << p.z << ", " << p.w <<")" << std::endl;
	}
};

#if 0
bool init_gl(void)
{
    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, g_window_width, g_window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)g_window_width / (GLfloat) g_window_height, 0.1, 10.0);

    return true;
} // end init_gl

void display(void)
{
    // transform the mesh
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last(g_mesh_width * g_mesh_height);

    thrust::transform(first, last,
                      g_vec.begin(),
                      sine_wave(g_mesh_width,g_mesh_height,g_anim));

    // map the vector into GL
    thrust::device_ptr<float4> ptr = &g_vec[0];

    // pass the device_ptr to the allocator's static function map_buffer
    // to map it into GL
    GLuint buffer = gl_vector::allocator_type::map_buffer(ptr);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, g_translate_z);
    glRotatef(g_rotate_x, 1.0, 0.0, 0.0);
    glRotatef(g_rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, g_mesh_width * g_mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();
    glutPostRedisplay();

    g_anim += 0.001;

    // unmap the vector from GL
    gl_vector::allocator_type::unmap_buffer(buffer);
} // end display

void mouse(int button, int state, int x, int y)
{
    if(state == GLUT_DOWN)
    {
	g_mouse_buttons |= 1<<button;
    } // end if
    else if(state == GLUT_UP)
    {
	g_mouse_buttons = 0;
    } // end else if

    g_mouse_old_x = x;
    g_mouse_old_y = y;
    glutPostRedisplay();
} // end mouse

void motion(int x, int y)
{
    float dx, dy;
    dx = x - g_mouse_old_x;
    dy = y - g_mouse_old_y;

    if(g_mouse_buttons & 1)
    {
	g_rotate_x += dy * 0.2;
	g_rotate_y += dx * 0.2;
    } // end if
    else if(g_mouse_buttons & 4)
    {
	g_translate_z += dy * 0.01;
    } // end else if

    g_mouse_old_x = x;
    g_mouse_old_y = y;
} // end motion

void keyboard(unsigned char key, int, int)
{
    switch(key)
    {
    // catch 'esc'
    case(27):
	      // deallocate memory
	      g_vec.clear();
    g_vec.shrink_to_fit();
    exit(0);
    default:
	break;
    } // end switch
} // end keyboard

int main(int argc, char** argv)
{
    // Create GL context
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(g_window_width, g_window_height);
    glutCreateWindow("Thrust/GL interop");

    // initialize GL
    if(!init_gl())
    {
	throw std::runtime_error("Couldn't initialize OpenGL");
    } // end if

    // register callbacks
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);

    // resize the vector to fit the mesh
    g_vec.resize(g_mesh_width * g_mesh_height);

    // transform the mesh
    thrust::counting_iterator<int,thrust::device_system_tag> first(0);
    thrust::counting_iterator<int,thrust::device_system_tag> last(g_mesh_width * g_mesh_height);

    thrust::transform(first, last,
                      g_vec.begin(),
                      sine_wave(g_mesh_width,g_mesh_height,g_anim));

    // start rendering mainloop
    glutMainLoop();

    return 0;
} // end main

#else
int main()
{
    sine_wave field(GRID_SIZE, GRID_SIZE);

    thrust::host_vector<thrust::tuple<int, int> > position(field.grid_coordinates_begin(), field.grid_coordinates_end());
    thrust::for_each(position.begin(), position.end(), print_tuple2());
    thrust::host_vector<float4> points(field.point_data_begin(), field.point_data_end());
    thrust::for_each(points.begin(), points.end(), print_float4());

    field.resize(2*GRID_SIZE, 2*GRID_SIZE);
    position.resize(thrust::distance(field.grid_coordinates_begin(), field.grid_coordinates_end()));
    points.resize(thrust::distance(field.grid_coordinates_begin(), field.grid_coordinates_end()));

    thrust::copy(field.grid_coordinates_begin(), field.grid_coordinates_end(), position.begin());
    thrust::for_each(position.begin(), position.end(), print_tuple2());
    thrust::copy(field.point_data_begin(), field.point_data_end(), points.begin());
    thrust::for_each(points.begin(), points.end(), print_float4());

    return 0;
}
#endif
