#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifdef __APPLE__
    #include <GL/glew.h>
    #include <OpenGL/OpenGL.h>
    #include <GLUT/glut.h>
#else
    #include <GL/glew.h>
    #include <GL/glut.h>
    #include <GL/gl.h>
#endif

#include <QtGui>
#include <QtOpenGL>
#include <QObject>

#include <cuda_gl_interop.h>

#include <vtkXMLImageDataReader.h>

#include <cutil_math.h>
#include <piston/choose_container.h>

#define SPACE thrust::device_system_tag
using namespace piston;

#include <piston/image3d.h>
#include <piston/vtk_image3d.h>
#include <piston/threshold_geometry.h>

#include <sys/time.h>
#include <stdio.h>
#include <math.h>

#include "glwindow.h"

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)

struct timeval begin, end, diff;
int frame_count = 0;
int grid_size = 256;
float cameraFOV = 60.0;
bool wireframe = false;

vtk_image3d<int, float, SPACE>* image;
threshold_geometry<vtk_image3d<int, float, SPACE> >* threshold;

GLuint quads_vbo[2];
struct hipGraphicsResource *quads_pos_res, *quads_normal_res, *quads_color_res;
unsigned int buffer_size;


void create_vbo()
{
    glGenBuffers(2, quads_vbo);
    int error;

    std::cout << "number of vertices: " << thrust::distance(threshold->vertices_begin(), threshold->vertices_end()) << std::endl;
    buffer_size = thrust::distance(threshold->vertices_begin(), threshold->vertices_end())* sizeof(float4);

    // initialize vertex buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_pos_res, quads_vbo[0],
                                              cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
      std::cout << "register pos buffer cuda error: " << error << "\n";
    }

    // initialize color buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if (hipGraphicsGLRegisterBuffer(&quads_color_res, quads_vbo[1],
                                     cudaGraphicsMapFlagsWriteDiscard) != hipSuccess) {
      std::cout << "register color buffer cuda error: " << error << "\n";
    }
}


GLWindow::GLWindow(QWidget *parent)
    : QGLWidget(QGLFormat(QGL::SampleBuffers), parent)
{
    setFocusPolicy(Qt::StrongFocus);
    timer = new QTimer(this);
    connect(timer, SIGNAL(timeout()), this, SLOT(updateGL()));
    timer->start(1);
}


GLWindow::~GLWindow()
{

}


QSize GLWindow::minimumSizeHint() const
{
    return QSize(100, 50);
}


QSize GLWindow::sizeHint() const
{
    return QSize(2048, 1024);
}


bool GLWindow::initialize(int argc, char *argv[])
{
    if (argc < 2) return false;
    sprintf(fileName, argv[1]);
    return true;
}


void GLWindow::initializeGL()
{
    glewInit();
    cudaGLSetGLDevice(0);

    vtkXMLImageDataReader *reader = vtkXMLImageDataReader::New();
    char fname[1024];
    sprintf(fname, "%s/%s", STRINGIZE_VALUE_OF(DATA_DIRECTORY), fileName);
    int fileFound = reader->CanReadFile(fname);
    if (fileFound == 0) sprintf(fname, fileName);
    reader->SetFileName(fname);
    reader->Update();

    vtkImageData *vtk_image = reader->GetOutput();

    image = new vtk_image3d<int, float, SPACE>(vtk_image);
    threshold = new threshold_geometry<vtk_image3d<int, float, SPACE> >(*image, -500.0f, -0.01f);
    (*threshold)();

    create_vbo();

    qrot.set(0,0,0,1);
    grid_size = vtk_image->GetDimensions()[1];

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    // good old-fashioned fixed function lighting
    float white[] = { 0.8, 0.8, 0.8, 1.0 };
    float lightPos[] = { 100.0, 100.0, -100.0, 1.0 };

    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);

    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, white);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_COLOR_MATERIAL);

    // Setup the view of the cube.
    glMatrixMode(GL_PROJECTION);
    gluPerspective( cameraFOV, 1.0, 1.0, grid_size*4.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt((grid_size-1)/2, (grid_size-1)/2, grid_size*1.5,
              (grid_size-1)/2, (grid_size-1)/2, 0.0,
              0.0, 1.0, 0.0);
}


void GLWindow::paintGL()
{
    timer->stop();

    if (frame_count == 0) gettimeofday(&begin, 0);

    (*threshold)();

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    if (wireframe) glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    else glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective( cameraFOV, 1.0, 1.0, grid_size*4.0);

    // set view matrix for 3D scene
    glMatrixMode(GL_MODELVIEW);
    glPushMatrix();

    qrot.getRotMat(rotationMatrix);
    glMultMatrixf(rotationMatrix);

    float3 center = make_float3((grid_size-1)/2, (grid_size-1)/2, 0.0);
    GLfloat matrix[16];
    glGetFloatv(GL_MODELVIEW_MATRIX, matrix);
    float3 offset = make_float3(matrix[0]*center.x + matrix[1]*center.y + matrix[2]*center.z, matrix[4]*center.x + matrix[5]*center.y + matrix[6]*center.z,
                                matrix[8]*center.x + matrix[9]*center.y + matrix[10]*center.z);
    offset.x = center.x - offset.x; offset.y = center.y - offset.y; offset.z = center.z - offset.z;
    glTranslatef(-offset.x, -offset.y, -offset.z);

    float4 *raw_ptr;
    size_t num_bytes;

    hipGraphicsMapResources(1, &quads_pos_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_pos_res);
    thrust::copy(thrust::make_transform_iterator(threshold->vertices_begin(), tuple2float4()),
                 thrust::make_transform_iterator(threshold->vertices_end(),   tuple2float4()),
                 thrust::device_ptr<float4>(raw_ptr));
    hipGraphicsUnmapResources(1, &quads_pos_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glVertexPointer(4, GL_FLOAT, 0, 0);
    glEnableClientState(GL_VERTEX_ARRAY);

    hipGraphicsMapResources(1, &quads_color_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_color_res);
    thrust::transform(threshold->scalars_begin(), threshold->scalars_end(),
                      thrust::device_ptr<float4>(raw_ptr),
                      color_map<float>(4.0f, 1600.0f));
    hipGraphicsUnmapResources(1, &quads_color_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glColorPointer(4, GL_FLOAT, 0, 0);
    glEnableClientState(GL_COLOR_ARRAY);

    glDrawArrays(GL_QUADS, 0, buffer_size/sizeof(float4));

    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    glPopMatrix();

    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    frame_count++;
    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    if (seconds > 0.5f)
    {
      char title[256];
      sprintf(title, "Marching Cube, fps: %2.2f", float(frame_count)/seconds);
      std::cout << title << std::endl;
      seconds = 0.0f;
      frame_count = 0;
    }

    timer->start(1);
}


void GLWindow::resizeGL(int width, int height)
{
    glViewport(0, 0, width, height);
}


void GLWindow::mousePressEvent(QMouseEvent *event)
{
    lastPos = event->pos();
}


void GLWindow::mouseMoveEvent(QMouseEvent *event)
{
    int dx = event->x() - lastPos.x();
    int dy = event->y() - lastPos.y();

    if (event->buttons() & Qt::LeftButton)
    {
      Quaternion newRotX;
      newRotX.setEulerAngles(-0.2*dx*3.14159/180.0, 0.0, 0.0);
      qrot.mul(newRotX);

      Quaternion newRotY;
      newRotY.setEulerAngles(0.0, 0.0, -0.2*dy*3.14159/180.0);
      qrot.mul(newRotY);
    }
    else if (event->buttons() & Qt::RightButton)
    {
      cameraFOV += dy/20.0;
    }
    lastPos = event->pos();
}


void GLWindow::keyPressEvent(QKeyEvent *event)
{
   if ((event->key() == 'w') || (event->key() == 'W'))
       wireframe = !wireframe;
}


