#include "hip/hip_runtime.h"
/*
 * marching_cube.cu
 *
 *  Created on: Sep 7, 2012
 *      Author: ollie
 */

#include <sys/time.h>

#include <vtkImageData.h>
#include <vtkRTAnalyticSource.h>

#include <piston/vtk_image3d.h>
#include "piston/marching_cube.h"

//#define SPACE thrust::host_system_tag
#define SPACE thrust::device_system_tag

using namespace piston;

int
main()
{
    vtkRTAnalyticSource *src = vtkRTAnalyticSource::New();
    src->SetWholeExtent(-100, 100, -100, 100, -100, 100);
    src->Update();

    vtk_image3d<SPACE> image(src->GetOutput());

    // get max and min of 3D scalars
    float min_iso = *thrust::min_element(image.point_data_begin(), image.point_data_end());
    float max_iso = *thrust::max_element(image.point_data_begin(), image.point_data_end());

    typedef vtk_image3d<SPACE> image_source;
    marching_cube<image_source, image_source> isosurface(image, image, 160.0f);

    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);
    for (float isovalue = min_iso; isovalue < max_iso; isovalue += ((max_iso-min_iso)/50)) {
	isosurface.set_isovalue(isovalue);
	isosurface();
    }
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);

    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    std::cout << "total time: " << seconds << ", fps: " << 50.f/seconds << std::endl;
    return 0;

}



