#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <sys/time.h>
#include <cmath>
#include <piston/util/cayley_field.h>
#include <piston/marching_cube.h>

#define GRID_SIZE 512
#define SPACE thrust::device_system_tag

using namespace piston;


int main(int argc, char **argv)
{
    cayley_field<SPACE> cayley(GRID_SIZE, GRID_SIZE, GRID_SIZE);

    // get max and min of 3D scalars
    float min_iso = *thrust::min_element(cayley.point_data_begin(), cayley.point_data_end());
    float max_iso = *thrust::max_element(cayley.point_data_begin(), cayley.point_data_end());

    // create a isosurface filter with cayley as input
    marching_cube<cayley_field<SPACE>, cayley_field<SPACE> > contour(cayley, cayley, 0.0f);

    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);
    for (float isovalue = min_iso; isovalue < max_iso; isovalue += ((max_iso-min_iso)/50)) {
	contour.set_isovalue(isovalue);
	contour();
    }
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    std::cout << "GRID_SIZE: " << GRID_SIZE << ", total time: " << seconds << std::endl;
    return 0;
}
